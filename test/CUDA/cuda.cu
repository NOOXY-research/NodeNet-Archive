#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include "hip/hip_runtime.h"
#define DATA_SIZE 1048576 * 128

int data[DATA_SIZE];
void GenerateNumbers(int *number, int size)
{
  srand((time(NULL)));
    for(int i = 0; i < size; i++) {
        number[i] = rand() % 10;
    }
}
bool InitCUDA()
{
    int count;

    hipGetDeviceCount(&count);
    if(count == 0) {
        fprintf(stderr, "There is no device.\n");
        return false;
    }

    int i;
    for(i = 0; i < count; i++) {
        hipDeviceProp_t prop;
        if(hipGetDeviceProperties(&prop, i) == hipSuccess) {
            if(prop.major >= 1) {
                break;
            }
        }
    }

    if(i == count) {
        fprintf(stderr, "There is no device supporting CUDA 1.x.\n");
        return false;
    }

    hipSetDevice(i);

    return true;
}
__global__ static void sumOfSquares(int *num, int* result, clock_t* time)
{
    int sum = 0;
    int i;
    clock_t start = clock();
    for(i = 0; i < DATA_SIZE; i++) {
        sum += num[i] * num[i];
    }

    *result = sum;
    *time = clock() - start;
}
int main()
{
    if(!InitCUDA()) {
        return 0;
    }

   printf("CUDA initialized.\n");
   GenerateNumbers(data, DATA_SIZE);
   int* gpudata, *result;
   clock_t* time;
  printf("start.\n");
   hipMalloc((void**) &gpudata, sizeof(int) * DATA_SIZE);
   hipMalloc((void**) &result, sizeof(int));
   hipMalloc((void**) &time, sizeof(clock_t));
   hipMemcpy(gpudata, data, sizeof(int) * DATA_SIZE, hipMemcpyHostToDevice);
   sumOfSquares<<<1, 1, 0>>>(gpudata, result, time);
   int sum;
   clock_t time_used;
   hipMemcpy(&sum, result, sizeof(int), hipMemcpyDeviceToHost);
   hipMemcpy(&time_used, time, sizeof(clock_t), hipMemcpyDeviceToHost);
   hipFree(gpudata);
   hipFree(result);
   printf("sum: %d time: %d\n", sum, time_used);

   sum = 0;
    for(int i = 0; i < DATA_SIZE; i++) {
        sum += data[i] * data[i];
    }
    printf("sum (CPU): %d\n", sum);
    return 0;
}
