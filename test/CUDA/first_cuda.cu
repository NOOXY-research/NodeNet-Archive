// First CUDA program


#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define DATA_SIZE	1048577
#define BLOCK_NUM	32
#define THREAD_NUM	256

int data[DATA_SIZE];


__global__ static void sumOfSquares(int *num, int* result, clock_t* time)
{
	extern __shared__ int shared[];
	const int tid = threadIdx.x;
	const int bid = blockIdx.x;
    int i;
	if(tid == 0) time[bid] = clock();
	shared[tid] = 0;

	for(i = bid * THREAD_NUM + tid; i < DATA_SIZE; i += BLOCK_NUM * THREAD_NUM) {
       shared[tid] += num[i] * num[i];
    }

    __syncthreads();

	if(tid < 128) { shared[tid] += shared[tid + 128]; } __syncthreads();
	if(tid < 64) { shared[tid] += shared[tid + 64]; } __syncthreads();
	if(tid < 32) { shared[tid] += shared[tid + 32]; } __syncthreads();
	if(tid < 16) { shared[tid] += shared[tid + 16]; } __syncthreads();
	if(tid < 8) { shared[tid] += shared[tid + 8]; } __syncthreads();
	if(tid < 4) { shared[tid] += shared[tid + 4]; } __syncthreads();
	if(tid < 2) { shared[tid] += shared[tid + 2]; } __syncthreads();
	if(tid < 1) { shared[tid] += shared[tid + 1]; } __syncthreads();

	if(tid == 0) {
		result[bid] = shared[0];
		time[bid + BLOCK_NUM] = clock();
	}
}


bool InitCUDA()
{
    int count;

    hipGetDeviceCount(&count);
    if(count == 0) {
       fprintf(stderr, "There is no device.\n");
       return false;
    }

    int i;
    for(i = 0; i < count; i++) {
       hipDeviceProp_t prop;
       if(hipGetDeviceProperties(&prop, i) == hipSuccess) {
          if(prop.major >= 1) {
             break;
          }
       }
    }

    if(i == count) {
       fprintf(stderr, "There is no device supporting CUDA 1.x.\n");
       return false;
    }

    hipSetDevice(i);

    return true;
}


void GenerateNumbers(int *number, int size)
{
    for(int i = 0; i < size; i++) {
       number[i] = rand() % 10;
    }
}


int main()
{
	if(!InitCUDA()) {
		return 0;
	}

	printf("CUDA initialized.\n");

	GenerateNumbers(data, DATA_SIZE);
printf("start.\n");
    int* gpudata, *result;
    clock_t* time;
    hipMalloc((void**) &gpudata, sizeof(int) * DATA_SIZE);
    hipMalloc((void**) &result, sizeof(int) * BLOCK_NUM);
    hipMalloc((void**) &time, sizeof(clock_t) * BLOCK_NUM * 2);
    hipMemcpy(gpudata, data, sizeof(int) * DATA_SIZE, hipMemcpyHostToDevice);

    sumOfSquares<<<BLOCK_NUM, THREAD_NUM, THREAD_NUM * sizeof(int)>>>(gpudata, result, time);

    int sum[BLOCK_NUM];
    clock_t time_used[BLOCK_NUM * 2];
    hipMemcpy(&sum, result, sizeof(int) * BLOCK_NUM, hipMemcpyDeviceToHost);
    hipMemcpy(&time_used, time, sizeof(clock_t) * BLOCK_NUM * 2, hipMemcpyDeviceToHost);
    hipFree(gpudata);
    hipFree(result);
	hipFree(time);

	int final_sum = 0;
	for(int i = 0; i < BLOCK_NUM; i++) {
		final_sum += sum[i];
	}

	clock_t min_start, max_end;
	min_start = time_used[0];
	max_end = time_used[BLOCK_NUM];
	for(int i = 1; i < BLOCK_NUM; i++) {
		if(min_start > time_used[i]) min_start = time_used[i];
		if(max_end < time_used[i + BLOCK_NUM]) max_end = time_used[i + BLOCK_NUM];
	}

    printf("sum: %d  time: %lu\n", final_sum, max_end - min_start);

    final_sum = 0;
    for(int i = 0; i < DATA_SIZE; i++) {
        final_sum += data[i] * data[i];
    }
    printf("sum (CPU): %d\n", final_sum);

	return 0;
}
